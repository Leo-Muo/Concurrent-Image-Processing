#include "hip/hip_runtime.h"
﻿#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#define BYTE_BOUND(value) value < 0 ? 0 : (value > 255 ? 255 : value)
#include <stdlib.h>
#include <chrono>
#include "stb_image.h"
#include <stdio.h>
#include <fstream>
#include <cstdint>
#include <cstdio>
#include <stdint.h>
#include "stb_image_write.h"

#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
#include ""


using namespace std;


__global__ void gaussian_blur(int* dev_ker_w, int* dev_ker_h, double* dev_kern, int* dev_channels, size_t* dev_size, int* dev_width, int* dev_height, unsigned char* dev_image, unsigned char* dev_output_image);
void grayscale(int channels, size_t size, unsigned char* data);


int main() {

    double ker3x3[9] = {
        1, 3, 1,
        3, 9, 3,
        1, 3, 1
    };

    double ker5x5[25] = {
       1,4,7,4,1,
       4,16,26,16,4,
       7,26,41,26,7,
       4,16,26,16,4,
       1,4,7,4,1
    };

    double ker7x7[49] = {
       1,4,7,10,7,4,1,
       4,12,26,33,26,12,4,
       7,26,55,71,55,26,7,
       10,33,71,91,71,33,10,
       7,26,55,71,55,26,7,
       4,12,26,33,26,12,4,
       1,4,7,10,7,4,1
    };

    int width, height, channels;
    
    int kernel_diamension = 7;

    /*if (img == NULL) {
        printf("Error in loading the image\n");
        exit(1);
    }*/

    //printf("Loaded image with a width of %dpx, a height of %dpx and %d channels\n", width, height, channels);

    double* dev_kern;
    int* dev_ker_w;
    int* dev_ker_h;
    int* dev_channels;
    int* dev_width;
    int* dev_height;
    size_t* dev_size;
    unsigned char* dev_image;
    unsigned char* dev_output_image = NULL;
    unsigned char* dev_data_temp = NULL;
    int j = 1;

    const char* image_file_array[8] = { "img1.jpg", "img2.jpg", "img3.jpeg", "img4.jpg", "img5.jpg", "img6.jpg", "img7.jpg", "img8.jpg" };
    
    for (int i = 1; i <= 32; i++)
    {
        unsigned char *img = stbi_load(image_file_array[i], &width, &height, &channels, 0);
       // std::cout << "Read: " << image_file_array[3] << endl;
        size_t size = width * height * channels;
        unsigned char* output_img = (unsigned char*)malloc(size * sizeof(unsigned char));
        unsigned char* data_temp = (unsigned char*)malloc(size * sizeof(unsigned char));

        auto start = chrono::steady_clock::now();
        hipMalloc(&dev_ker_w, sizeof(int));
        hipMalloc(&dev_ker_h, sizeof(int));
        hipMalloc(&dev_channels, sizeof(int));
        hipMalloc(&dev_width, sizeof(int));
        hipMalloc(&dev_height, sizeof(int));
        hipMalloc(&dev_size, sizeof(size_t));
        hipMalloc((void**)&dev_image, size * sizeof(unsigned char));
        hipMalloc((void**)&dev_output_image, size * sizeof(unsigned char));
        hipMalloc(&dev_kern, 49 * sizeof(double));

        hipMemcpy(dev_ker_w, &kernel_diamension, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(dev_ker_h, &kernel_diamension, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(dev_channels, &channels, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(dev_width, &width, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(dev_height, &height, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(dev_size, &size, sizeof(size_t), hipMemcpyHostToDevice);
        hipMemcpy(dev_kern, &ker7x7, 49 * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(dev_image, img, size * sizeof(unsigned char), hipMemcpyHostToDevice);

        gaussian_blur << <i, channels >> > (dev_ker_w, dev_ker_h, dev_kern, dev_channels, dev_size, dev_width, dev_height, dev_image, dev_output_image);
        hipDeviceSynchronize();
        hipMemcpy(output_img, dev_output_image, size * sizeof(unsigned char), hipMemcpyDeviceToHost);

        auto end = chrono::steady_clock::now();
        auto time = chrono::duration_cast<chrono::milliseconds>(end - start).count();
        //std::cout << "Elapsed time in milliseconds: " << time << " ms" << endl;
       // std::cout << "=========================================================" << endl;

       // stbi_write_png("result.png", width, height, channels, output_img, width * channels);

        int fileNumber = 0;
        
        std::ofstream myfile;
        myfile.open("results.csv", std::ios_base::app);
        myfile << time << "," << i << endl;
        myfile.close();



    }

    

    hipFree(dev_ker_w);
    hipFree(dev_ker_h);
    hipFree(dev_channels);
    hipFree(dev_width);
    hipFree(dev_height);
    hipFree(dev_size);
    hipFree(dev_image);
    hipFree(dev_output_image);

    cout << "Finished" << endl;
}


__global__ void gaussian_blur(int* dev_ker_w, int* dev_ker_h, double* dev_kern, int* dev_channels,
    size_t* dev_size, int* dev_width, int* dev_height, unsigned char* dev_image, unsigned char* dev_output_image) {

    const auto num_row = *dev_height / gridDim.x;

    if (num_row >= 1) {

        int cr = (*dev_ker_w - 1) / 2;
        int cc = (*dev_ker_h - 1) / 2;
        int center = cr * (*dev_ker_w) + cc;


        int dmo = 0;
        for (int i = 0; i < ((*dev_ker_h) * (*dev_ker_w)); i++)
        {
            dmo += dev_kern[i];
        }


        unsigned long start_row = num_row * blockIdx.x;
        unsigned long end_row;
        if (blockIdx.x == gridDim.x - 1)
        {
             end_row = *dev_height;
        }
        else {
             end_row = num_row * (blockIdx.x + 1);
        }
        

        int channel = threadIdx.x;
        int k = (start_row * (*dev_width) ) + channel;
        while ( k <= ((*dev_width) * end_row * (*dev_channels)) - 1)
        {
            double sum = 0;

            for (long i = -((long)cr); i < (long)*dev_ker_h - cr; ++i) {
                long row = ((long)k / (*dev_channels)) / (*dev_width) - i;
                if (row < 0 || row >(*dev_height) - 1) {
                    continue;
                }

                for (long j = -((long)cc); j < (long)*dev_ker_w - cc; ++j) {
                    long col = ((long)k / (*dev_channels)) % (*dev_width) - j;
                    if (col < 0 || col >(*dev_width) - 1) {
                        continue;
                    }
                    sum += dev_kern[center + i * (long)*dev_ker_w + j] * dev_image[(row * (*dev_width) + col) * (*dev_channels) + channel];
                }
            }
            dev_output_image[k] = (unsigned char)BYTE_BOUND(round(sum / dmo));
            k += *dev_channels; 
        }
    }
    else {
        printf("Number of BLocks are large\n");
    }
}






void grayscale(int channels, size_t size, unsigned char* data) {
    if (channels < 3)
    {
        std::cout << "Image has less than 3 channels" << endl;
    }
    else {
        for (int i = 0; i < size; i += channels)
        {
            int gray = (data[i] + data[i + 1] + data[i + 2]) / 3;
            memset(data + i, gray, 3);
        }
    }

}