#include "hip/hip_runtime.h"
﻿#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#define BYTE_BOUND(value) value < 0 ? 0 : (value > 255 ? 255 : value)
#include <stdlib.h>
#include <chrono>
#include "stb_image.h"
#include <stdio.h>
#include <fstream>
#include <cstdint>
#include <cstdio>
#include <stdint.h>
#include "stb_image_write.h"

#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
#include ""


using namespace std;

__global__ void run_on_gpu();
__device__ void execute(const char* name);
void grayscale(int channels, size_t size, unsigned char* data);



int main() {
    int width, height, channels;
    unsigned char* img = stbi_load("img1.jpg", &width, &height, &channels, 0);
    size_t size = width * height * channels;

    if (img == NULL) {
        printf("Error in loading the image\n");
        exit(1);
    }

    printf("Loaded image with a width of %dpx, a height of %dpx and %d channels\n", width, height, channels);

    grayscale(channels, size, img);

    stbi_write_png("sky.png", width, height, channels, img, width * channels);
    stbi_write_jpg("sky2.jpg", width, height, channels, img, 100);

    run_on_gpu << <1, 2 >> > ();
    hipDeviceSynchronize();
    cout << "Finished" << endl;
}

void grayscale(int channels, size_t size, unsigned char* data) {
    if (channels < 3)
    {
        std::cout << "Image has less than 3 channels" << endl;
    }
    else {
        for (int i = 0; i < size; i += channels)
        {
            int gray = (data[i] + data[i + 1] + data[i + 2]) / 3;
            memset(data + i, gray, 3);
        }
    }
   
}

__global__ void run_on_gpu() {
    const char* name;
    if (threadIdx.x == 0) {
        name = "Thread 1";
    }
    else {
        name = "Thread 2";
    }
    execute(name);
}

__device__ void execute(const char* name) {
    printf("%s: first\n", name);
    printf("%s: second\n", name);
    printf("%s: third\n", name);
}
